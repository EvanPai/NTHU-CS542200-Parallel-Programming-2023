#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 32

// Blocking factor（一個floyd warshall block要切多大）
#define B 64
#define offset 32
#include <sys/mman.h>
#include <sys/stat.h> 
#include <sys/types.h>
#include <fcntl.h>
#include <unistd.h>

//======================
#define DEV_NO 0
//cudaDeviceProp prop;

const int INF = ((1 << 30) - 1);
//const int V = 50010;
int n, m;
int N; // 用來算GPU memory size

//可優化，讓Dist的大小根據input決定
int *Dist = NULL;
//static int Dist[V][V];

// 可以加上inline來增快
void input(char* infile);
void output(char* outFileName);
int ceil(int a, int b);
void block_FW();

__global__ void phase1(int *dst, int Round, int N);
__global__ void phase2_1(int *dst, int Round, int N);
__global__ void phase2_2(int *dst, int Round, int N);
__global__ void phase3(int *dst, int Round, int N);


void input(char* infile) {
	int file = open(infile, O_RDONLY);
    int a = 0;
	int *ft = (int*)mmap(NULL, 2*sizeof(int), PROT_READ, MAP_PRIVATE, file, 0);
    m = ft[1];
    n = ft[0];
    // n是有幾個vertex, m是有幾個edge
    // 設定N，之後kernel計算就不用branch來看boundry
	if (n % B) N = n + (B - n % B);
	else N = n;

    int *pair = (int*)(mmap(NULL, (3 * m + 2) * sizeof(int), PROT_READ, MAP_PRIVATE, file, 0));
	Dist = (int*)malloc(N*N*sizeof(int));

	for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            if (i == j) {
                Dist[i*N + j] = 0;
            } else {
                Dist[i*N + j] = INF;
            }
        }
    }

	#pragma unroll 4
	for (int i = 0; i < m; ++i) {
		Dist[pair[i*3+2]*N+pair[i*3+3]]= pair[i*3+4];
	}

	close(file);
	munmap(pair, (3 * m + 2) * sizeof(int));
}

void output(char* outFileName) {
    FILE* outfile = fopen(outFileName, "w");
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            if (Dist[i*N + j] >= INF) Dist[i*N + j] = INF;
        }
        fwrite(&Dist[i*N], sizeof(int), n, outfile);
    }
    fclose(outfile);
}

int ceil(int a, int b) { return (a + b - 1) / b; }

void block_FW() {
	int round = ceil(n, B);
	int *dst = NULL;
	unsigned int size = N*N*sizeof(int);

    // 把Dist memory pin住，增加performance
	hipHostRegister(Dist, size, hipHostRegisterDefault);

    // 在GPU中開一塊size大小的memory給dst
	hipMalloc(&dst, size);

    // 把Dist搬進GPU中的dst裡面
	hipMemcpy(dst, Dist, size, hipMemcpyHostToDevice);
	
    // 總共要 N / B個blocks（包含有多出來的）
	int blocks = (N + B - 1) / B;
	
    dim3 block_dim(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid_dim(blocks, blocks);


    // Create CUDA streams
    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
	for (int r = 0; r < round; ++r) {
		// phase 1
		phase1<<<1, block_dim>>>(dst, r, N);
		
		// phase 2
		phase2_1<<<blocks, block_dim, 0, stream1>>>(dst, r, N);
        phase2_2<<<blocks, block_dim, 0, stream2>>>(dst, r, N);

		// Synchronize with both streams
        hipStreamSynchronize(stream1);
        hipStreamSynchronize(stream2);

		// phase 3
		phase3<<<grid_dim, block_dim>>>(dst, r, N);
	}

    // GPU算完搬回CPU
	hipMemcpy(Dist, dst, size, hipMemcpyDeviceToHost);

    // 清掉dst
	hipFree(dst);
}


// ------------------------ GPU --------------------------- //
__device__ int Min(int a, int b) {
	return min(a, b);
}

__global__ void phase1(int *dst, int Round, int N) {
    int y = threadIdx.y; // y軸 = row
	int y_offset = y + offset;
	__shared__ int s[B][B];
	int x = threadIdx.x; // x軸 = column
	int x_offset = x + offset;

	// y => 0~31
    // x => 0~31

    

    // 因為最多只能用1024(32 * 32)個threads，但要算(64 * 64)大小的block(B=64)
    // 且又要盡量用shared memory
    // 所以讓一個thread算4個點的資料。
	int top_left = Round * B * (N + 1) + y * N + x;
    s[y][x] = dst[top_left];

	int top_right = Round * B * (N + 1) + y * N + x + offset;
	s[y][x_offset] = dst[top_right];

	int bottom_left = Round * B * (N + 1) + (y + offset) * N + x;
	s[y_offset][x] = dst[bottom_left];

	int bottom_right = Round * B * (N + 1) + (y + offset) * N + x + offset;
	s[y_offset][x_offset] = dst[bottom_right];
	// load gloabal data to shared memory
	
	__syncthreads();

	for (int k = 0; k < B; ++k) {
		s[y][x] = Min(s[y][k] + s[k][x], s[y][x]);
		s[y][x_offset] = Min(s[y][k] + s[k][x_offset], s[y][x_offset]);
		s[y_offset][x] = Min(s[y_offset][k] + s[k][x], s[y_offset][x]);
		s[y_offset][x_offset] = Min(s[y_offset][k] + s[k][x_offset], s[y_offset][x_offset]);
		__syncthreads();
	}
	dst[top_left] = s[y][x];
	dst[top_right] = s[y][x_offset];
	dst[bottom_left] = s[y_offset][x];
	dst[bottom_right] = s[y_offset][x_offset];
}

__global__ void phase2_1(int *dst, int Round, int N) {
	if (blockIdx.x == Round) return;
	__shared__ int s[B][B];
	int y = threadIdx.y;
	int y_B = y + offset;
	__shared__ int col[B][B];
	int x = threadIdx.x;
	int x_B = x + offset;

	
	

    // 要算跟pivot B有row或col相同的所有B
    // 一樣，每個thread要算4個點
    // 算col的matrix B
    
	int main_top_left = Round * B * (N + 1) + y * N + x;
    s[y][x] = dst[main_top_left];
    int col_top_left = blockIdx.x * B * N + Round * B + y * N + x;
	col[y][x] = dst[col_top_left];

	int main_top_right = Round * B * (N + 1) + y * N + x + offset;
    s[y][x_B] = dst[main_top_right];
	int col_top_right = blockIdx.x * B * N + Round * B + y * N + x + offset;
	col[y][x_B] = dst[col_top_right];


	int main_bottom_left = Round * B * (N + 1) + (y + offset) * N + x;
    s[y_B][x] = dst[main_bottom_left];
	int col_bottom_left = blockIdx.x * B * N + Round * B + (y + offset) * N + x;
	col[y_B][x] = dst[col_bottom_left];


	int main_bottom_right = Round * B * (N + 1) + (y + offset) * N + x + offset;
    s[y_B][x_B] = dst[main_bottom_right];
	int col_bottom_right = blockIdx.x * B * N + Round * B + (y + offset) * N + x + offset;
	col[y_B][x_B] = dst[col_bottom_right];

	__syncthreads();
	
	for (int k = 0; k < B; ++k) {
        col[y][x] = Min(col[y][x], col[y][k] + s[k][x]);
        col[y][x_B] = Min(col[y][x_B], col[y][k] + s[k][x_B]);
        col[y_B][x] = Min(col[y_B][x], col[y_B][k] + s[k][x]);
        col[y_B][x_B] = Min(col[y_B][x_B], col[y_B][k] + s[k][x_B]);
		__syncthreads();
	}
    dst[col_top_left] = col[y][x];
    dst[col_top_right] = col[y][x_B];
    dst[col_bottom_left] = col[y_B][x];
    dst[col_bottom_right] = col[y_B][x_B];
}

__global__ void phase2_2(int *dst, int Round, int N) {
	if (blockIdx.x == Round) return;
	__shared__ int s[B][B];
	int y = threadIdx.y;
	int y_B = y + offset;
	__shared__ int row[B][B];
	int x = threadIdx.x;
	int x_B = x + offset;

    // 要算跟pivot B有row或col相同的所有B
    // 一樣，每個thread要算4個點

    // 算row的matrix B

	int main_top_left = Round * B * (N + 1) + y * N + x;
    s[y][x] = dst[main_top_left];
    int row_top_left = Round * B * N + blockIdx.x * B + y * N + x;
    row[y][x] = dst[row_top_left];

	int main_top_right = Round * B * (N + 1) + y * N + x + offset;
    s[y][x_B] = dst[main_top_right];
    int row_top_right = Round * B * N + blockIdx.x * B + y * N + x + offset;
    row[y][x_B] = dst[row_top_right];

	int main_bottom_left = Round * B * (N + 1) + (y + offset) * N + x;
    s[y_B][x] = dst[main_bottom_left];
    int row_bottom_left = Round * B * N + blockIdx.x * B + (y + offset) * N + x;
    row[y_B][x] = dst[row_bottom_left];

	int main_bottom_right = Round * B * (N + 1) + (y + offset) * N + x + offset;
    s[y_B][x_B] = dst[main_bottom_right];
    int row_bottom_right = Round * B * N + blockIdx.x * B + (y + offset) * N + x + offset;
    row[y_B][x_B] = dst[row_bottom_right];

	__syncthreads();
	
	for (int k = 0; k < B; ++k) {
        row[y][x] = Min(row[y][x], s[y][k] + row[k][x]);
        row[y][x_B] = Min(row[y][x_B], s[y][k] + row[k][x_B]);
        row[y_B][x] = Min(row[y_B][x], s[y_B][k] + row[k][x]);
        row[y_B][x_B] = Min(row[y_B][x_B], s[y_B][k] + row[k][x_B]);
		__syncthreads();
	}
    dst[row_top_left] = row[y][x];
    dst[row_top_right] = row[y][x_B];
    dst[row_bottom_left] = row[y_B][x];
    dst[row_bottom_right] = row[y_B][x_B];
}

__global__ void phase3(int *dst, int Round, int N) {
	if (blockIdx.x == Round || blockIdx.y == Round) return;
	__shared__ int col[B][B];
	int y = threadIdx.y;
	int y_B = y + offset;
	__shared__ int row[B][B];
	int x = threadIdx.x;
	int x_B = x + offset;
	__shared__ int target[B][B];
    
    
    

    int target_top_left = blockIdx.y * B * N + blockIdx.x * B + y * N + x;
    target[y][x] = dst[target_top_left];
    int col_top_left = blockIdx.y * B * N + Round * B + y * N + x;
    col[y][x] = dst[col_top_left];
    int row_top_left = Round * B * N + blockIdx.x * B + y * N + x;
    row[y][x] = dst[row_top_left];

    int target_top_right = blockIdx.y * B * N + blockIdx.x * B + y * N + x + offset;
    target[y][x_B] = dst[target_top_right];
    int col_top_right = blockIdx.y * B * N + Round * B + y * N + x + offset;
    col[y][x_B] = dst[col_top_right];
    int row_top_right = Round * B * N + blockIdx.x * B + y * N + x + offset;
    row[y][x_B] = dst[row_top_right];
	
    int target_bottom_left = blockIdx.y * B * N + blockIdx.x * B + (y + offset) * N + x;
    target[y_B][x] = dst[target_bottom_left];
    int col_bottom_left = blockIdx.y * B * N + Round * B + (y + offset) * N + x;
    col[y_B][x] = dst[col_bottom_left];
    int row_bottom_left = Round * B * N + blockIdx.x * B + (y + offset) * N + x;
    row[y_B][x] = dst[row_bottom_left];

    int target_bottom_right = blockIdx.y * B * N + blockIdx.x * B + (y + offset) * N + x + offset;
    target[y_B][x_B] = dst[target_bottom_right];
	int col_bottom_right = blockIdx.y * B * N + Round * B + (y + offset) * N + x + offset;
    col[y_B][x_B] = dst[col_bottom_right];
    int row_bottom_right = Round * B * N + blockIdx.x * B + (y + offset) * N + x + offset;
	row[y_B][x_B] = dst[row_bottom_right];
	
	__syncthreads();

	#pragma unroll 32
	for (int k = 0; k < B; ++k) {
		target[y][x] = Min(col[y][k] + row[k][x], target[y][x]);
		target[y][x_B] = Min(col[y][k] + row[k][x_B], target[y][x_B]);
		target[y_B][x] = Min(col[y_B][k] + row[k][x], target[y_B][x]);
		target[y_B][x_B] = Min(col[y_B][k] + row[k][x_B], target[y_B][x_B]);
	}
	dst[target_top_left] = target[y][x];
	dst[target_top_right] = target[y][x_B];
	dst[target_bottom_left] = target[y_B][x];
	dst[target_bottom_right] = target[y_B][x_B];
}

int main(int argc, char* argv[]) {
    input(argv[1]);
    
    //cudaGetDeviceProperties(&prop, DEV_NO);
    //printf("maxThreasPerBlock = %d, sharedMemPerBlock = %d", prop.maxThreasPerBlock, prop.sharedMemPerBlock);

    block_FW();
    output(argv[2]);
    return 0;
}